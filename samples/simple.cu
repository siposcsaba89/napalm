#include "hip/hip_runtime.h"
#include <sm_32_intrinsics.h>
#include <surface_functions.h>
//#include <sample_inc.h>
 
extern "C"
{
    __global__ void test_kernel( const unsigned char * src, unsigned char * dst, int multiplier)
    {
        dst[threadIdx.x + blockIdx.x * blockDim.x] = src[threadIdx.x + blockIdx.x * blockDim.x] * multiplier;
    } 
}