
#include <hip/hip_runtime.h>


extern "C"
{
    __global__ void test_kernel( const unsigned char * src, unsigned char * dst, int multiplier)
    {
        dst[threadIdx.x + blockIdx.x * blockDim.x] = src[threadIdx.x + blockIdx.x * blockDim.x] * multiplier;
    }
}